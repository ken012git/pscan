#include "hip/hip_runtime.h"
#include <torch/extension.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipcub/hipcub.hpp>
#include <hip/hip_fp16.h>
#include <ATen/cuda/HIPContext.h>

#include <vector>
#include "pscan_cuda_v1.cuh"

#include <iostream>
#include <stdio.h>

#define CHECK_CUDA_ERROR(val) check((val), #val, __FILE__, __LINE__)
template <typename T>
void check(T err, const char *const func, const char *const file,
           const int line)
{
    if (err != hipSuccess)
    {
        std::cerr << "CUDA Runtime Error at: " << file << ":" << line
                  << std::endl;
        std::cerr << hipGetErrorString(err) << " " << func << std::endl;
        std::exit(EXIT_FAILURE);
    }
}

template <typename T>
struct PairScalar;

template <>
struct PairScalar<int8_t>
{
    typedef char2 type;
};

template <>
struct PairScalar<int>
{
    typedef int2 type;
};

template <>
struct PairScalar<at::Half>
{
    typedef half2 type;
};

template <>
struct PairScalar<float>
{
    typedef float2 type;
};

template <>
struct PairScalar<double>
{
    typedef double2 type;
};

template <typename vec_t>
struct MultAddFunctor
{
    __device__ __forceinline__
        vec_t
        operator()(const vec_t &a, const vec_t &b) const
    {
        // return {a.x * b.x, a.y * b.x + b.y};
        return {__hmul(a.x, b.x), __hadd(__hmul(a.y, b.x), b.y)};
    }
};

template <typename scalar_t, int n, int m>
__device__ __forceinline__ void transpose(scalar_t *A)
{
    if (n == 2 && m == 2)
    {
        scalar_t tmp = A[1];
        A[1] = A[2];
        A[2] = tmp;
    }
    else if (n == 4 && m == 2)
    {
        scalar_t tmp = A[1];
        A[1] = A[4];
        A[4] = A[2];
        A[2] = tmp;
        tmp = A[3];
        A[3] = A[5];
        A[5] = A[6];
        A[6] = tmp;
    }
}

template <
    typename scalar_t,
    int ITEMS_PER_THREAD,
    int BLOCK_THREADS,
    bool REVERSE>
__global__ void pscan_cuda_forward_kernel(
    scalar_t *A,
    scalar_t *X,
    int dim_size,
    int state_size)
{
    // block ID
    const int bidx = blockIdx.x;
    const int didx = blockIdx.y;
    const int tid = threadIdx.x;
    int offset;
    if constexpr(REVERSE){
        offset = bidx * dim_size * state_size + didx * state_size + (BLOCK_THREADS - tid) * ITEMS_PER_THREAD - 1;
    }
    else{
        offset = bidx * dim_size * state_size + didx * state_size + tid * ITEMS_PER_THREAD;
    }

    typedef typename PairScalar<scalar_t>::type pair_type;
    typedef hipcub::BlockScan<pair_type, BLOCK_THREADS> BlockScanT;
    using TempStorageT = typename BlockScanT::TempStorage;

    extern __shared__ char smem[];

    auto &temp_storage = reinterpret_cast<TempStorageT &>(smem);

    pair_type thread_data[ITEMS_PER_THREAD];
    scalar_t* thread_data_scalar = reinterpret_cast<scalar_t*>(thread_data);

    if constexpr(REVERSE){
        if ((tid+1) * ITEMS_PER_THREAD <= state_size){
            #pragma unroll
            for (int i = 0; i < ITEMS_PER_THREAD; ++i){
                thread_data_scalar[i] = A[offset - i];
            }
            #pragma unroll
            for (int i = 0; i < ITEMS_PER_THREAD; ++i){
                thread_data_scalar[i+ITEMS_PER_THREAD] = X[offset - i];
            }
        }
    }
    else{
        if ((tid+1) * ITEMS_PER_THREAD <= state_size){
            #pragma unroll
            for (int i = 0; i < ITEMS_PER_THREAD; ++i){
                thread_data_scalar[i] = A[offset + i];
            }
            #pragma unroll
            for (int i = 0; i < ITEMS_PER_THREAD; ++i){
                thread_data_scalar[i+ITEMS_PER_THREAD] = X[offset + i];
            }
        }
    }


    // Inplace transpose of thread_data for small fixed size
    transpose<scalar_t, ITEMS_PER_THREAD, 2>(thread_data_scalar);
    BlockScanT(temp_storage).InclusiveScan(thread_data, thread_data, MultAddFunctor<pair_type>());
    transpose<scalar_t, ITEMS_PER_THREAD, 2>(thread_data_scalar);

    if constexpr(REVERSE){
        if ((tid+1) * ITEMS_PER_THREAD <= state_size){
            /*#pragma unroll
            for (int i = 0; i < ITEMS_PER_THREAD; ++i){
                A[offset - i] = thread_data_scalar[i];
            }*/
            #pragma unroll
            for (int i = 0; i < ITEMS_PER_THREAD; ++i){
                X[offset - i] = thread_data_scalar[i+ITEMS_PER_THREAD];
            }
        }
    }
    else{
        if ((tid+1) * ITEMS_PER_THREAD <= state_size){
            #pragma unroll
            for (int i = 0; i < ITEMS_PER_THREAD; ++i){
                A[offset + i] = thread_data_scalar[i];
            }
            #pragma unroll
            for (int i = 0; i < ITEMS_PER_THREAD; ++i){
                X[offset + i] = thread_data_scalar[i+ITEMS_PER_THREAD];
            }
        }
    }
}
template <typename T, int BLOCK_THREADS, int ARCH>
constexpr std::size_t arch_bytes_size = sizeof(typename hipcub::BlockScan<T,BLOCK_THREADS,hipcub::BLOCK_SCAN_RAKING ,1,1,ARCH>::TempStorage);
template <typename T, int BLOCK_THREADS, int... Archs>
constexpr auto archs_max_bytes = (std::max)(
    {
        arch_bytes_size<T, BLOCK_THREADS, Archs>...,
    });

const int TILE_DIM = 32;
const int BLOCK_ROWS = 8;

template <typename scalar_t>
__global__ void transposeNoBankConflicts(scalar_t *odata, const scalar_t *idata, const int stride)
{
    __shared__ scalar_t tile[TILE_DIM][TILE_DIM + 1]; // [32][33]

    int offset = blockIdx.z * stride; // z
    int x = blockIdx.x * TILE_DIM + threadIdx.x; // x * 32 + x
    int y = blockIdx.y * TILE_DIM + threadIdx.y; // y * 32 + y
    int width = gridDim.x * TILE_DIM; // 8*32=256
    int height = gridDim.y * TILE_DIM; // 64*32=2048

    // printf("(%d, %d) | (%d, %d, %d), (%d, %d)\n", gridDim.x, gridDim.y, blockIdx.x, blockIdx.y, blockIdx.z, threadIdx.x, threadIdx.y);

    if (x < width)
    {
#pragma unroll
        for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
        {
            if ((y + j) < height)
            {
                tile[threadIdx.y + j][threadIdx.x] = idata[offset + ((y + j) * width) + x];
            }
        }
    }

    __syncthreads();

    x = blockIdx.y * TILE_DIM + threadIdx.x; // transpose block offset
    y = blockIdx.x * TILE_DIM + threadIdx.y;

    if (x < height)
    {
#pragma unroll
        for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
        {
            if ((y + j) < width)
            {
                odata[offset + ((y + j) * height) + x] = tile[threadIdx.x][threadIdx.y + j];
            }
        }
    }
}

template <bool REVERSE>
torch::Tensor pscan_cuda_wrapper(torch::Tensor A, torch::Tensor X)
{
    // std::cout << "pscan_cuda_wrapper" << std::endl;
    // std::cout << "X.stride(0): " << X.stride(0) << std::endl;
    /*
    deltaA.shape:  torch.Size([1, 1536, 14, 16])
    x.shape:  torch.Size([1, 1536, 16])                                                                        
    deltaB_u.shape:  torch.Size([1, 1536, 14, 16])
    u.shape:  torch.Size([1, 1536, 14]) 14
    */
    // A [bsize, dim, seqlen], X [bsize, seqlen, dim]
    // torch.Size([384, 256, 2048]) torch.Size([384, 2048, 256])
    const auto batch_size = A.size(0); // bsize = 384
    const auto state_size = A.size(2); // seqlen  = 2048
    const auto dim_size = A.size(1);   // x dim = 256

    size_t const num_streams{4};
    const int offset = (batch_size / num_streams) * state_size * dim_size; // (384 / 4) * 2048 * 256 = 50331648

    std::vector<hipStream_t> streams(num_streams);
    torch::Tensor X_ = torch::empty({X.size(0), X.size(2), X.size(1)}, X.options()); // [bsize, dim, seqlen] = [384, 256, 2048]

    for (size_t i = 0; i < num_streams; ++i){
        CHECK_CUDA_ERROR(hipStreamCreate(&streams[i]));
    }

    /* --------------- All type (Not working) --------------- */
    // AT_DISPATCH_ALL_TYPES(A.type(), "pscan_transpose_cuda", ([&]
    //                                                               {
    //     for (size_t i = 0; i < num_streams; ++i){
    //         dim3 dimGrid(dim_size/TILE_DIM, state_size/TILE_DIM, batch_size / num_streams); // (8. 64, 96),  256 / 32, 2048 / 32, 384 / 4
    //         dim3 dimBlock(TILE_DIM, BLOCK_ROWS, 1); // 32, 8, 1
    //         transposeNoBankConflicts<<<dimGrid, dimBlock, 0, streams[i]>>>(
    //             X_.data<scalar_t>()+ offset*i,
    //             X.data<scalar_t>() + offset*i,
    //             X.stride(0)
    //         );
    //     } }));

    // const int threads = 1024;
    // const int elements_per_thread = 2;

    // AT_DISPATCH_ALL_TYPES(A.type(), "pscan_forward_cuda", ([&]
    //                                                             {
    // for(size_t i = 0; i < num_streams; ++i){
    //     const auto blocks = dim3(batch_size / num_streams, dim_size, 1);

    //     typedef typename PairScalar<scalar_t>::type pair_type;
    //     auto block_scan_temp_bytes = archs_max_bytes<pair_type, threads, 700, 800, 860>;
    //     auto smem_size = (std::max)(1 * sizeof(pair_type), block_scan_temp_bytes);
    
    //     pscan_cuda_forward_kernel<scalar_t, elements_per_thread, threads, REVERSE><<<blocks, threads, smem_size, streams[i]>>>(
    //         A.data<scalar_t>() + offset*i,
    //         X_.data<scalar_t>() + offset*i,
    //         dim_size,
    //         state_size
    //     );
    // } }));



    /* --------------- FP32 --------------- */
    // AT_DISPATCH_FLOATING_TYPES(A.type(), "pscan_transpose_cuda", ([&]
    //                                                               {
    //     for (size_t i = 0; i < num_streams; ++i){
    //         dim3 dimGrid(dim_size/TILE_DIM, state_size/TILE_DIM, batch_size / num_streams); // (8. 64, 96),  256 / 32, 2048 / 32, 384 / 4
    //         dim3 dimBlock(TILE_DIM, BLOCK_ROWS, 1); // 32, 8, 1
    //         transposeNoBankConflicts<<<dimGrid, dimBlock, 0, streams[i]>>>(
    //             X_.data<scalar_t>()+ offset*i,
    //             X.data<scalar_t>() + offset*i,
    //             X.stride(0)
    //         );
    //     } }));

    // const int threads = 1024;
    // const int elements_per_thread = 2;

    // AT_DISPATCH_FLOATING_TYPES(A.type(), "pscan_forward_cuda", ([&]
    //                                                             {
    // for(size_t i = 0; i < num_streams; ++i){
    //     const auto blocks = dim3(batch_size / num_streams, dim_size, 1);

    //     typedef typename PairScalar<scalar_t>::type pair_type;
    //     auto block_scan_temp_bytes = archs_max_bytes<pair_type, threads, 700, 800, 860>;
    //     auto smem_size = (std::max)(1 * sizeof(pair_type), block_scan_temp_bytes);
    
    //     pscan_cuda_forward_kernel<scalar_t, elements_per_thread, threads, REVERSE><<<blocks, threads, smem_size, streams[i]>>>(
    //         A.data<scalar_t>() + offset*i,
    //         X_.data<scalar_t>() + offset*i,
    //         dim_size,
    //         state_size
    //     );
    // } }));


    /* --------------- FP16 --------------- */
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(A.type(), "pscan_transpose_cuda", ([&]
                                                                  {
        for (size_t i = 0; i < num_streams; ++i){
            dim3 dimGrid(dim_size/TILE_DIM, state_size/TILE_DIM, batch_size / num_streams); // (8. 64, 96),  256 / 32, 2048 / 32, 384 / 4
            dim3 dimBlock(TILE_DIM, BLOCK_ROWS, 1); // 32, 8, 1
            transposeNoBankConflicts<<<dimGrid, dimBlock, 0, streams[i]>>>(
                X_.data<scalar_t>()+ offset*i,
                X.data<scalar_t>() + offset*i,
                X.stride(0)
            );
        } }));

    const int threads = 1024;
    const int elements_per_thread = 2;

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(A.type(), "pscan_forward_cuda", ([&]
                                                                {
    for(size_t i = 0; i < num_streams; ++i){
        const auto blocks = dim3(batch_size / num_streams, dim_size, 1);

        typedef typename PairScalar<at::Half>::type pair_type;
        auto block_scan_temp_bytes = archs_max_bytes<pair_type, threads, 700, 800, 860>;
        auto smem_size = (std::max)(1 * sizeof(pair_type), block_scan_temp_bytes);
    
        pscan_cuda_forward_kernel<at::Half, elements_per_thread, threads, REVERSE><<<blocks, threads, smem_size, streams[i]>>>(
            A.data<at::Half>() + offset*i,
            X_.data<at::Half>() + offset*i,
            dim_size,
            state_size
        );
    } }));


    /* --------------- INT8 --------------- */
    // AT_DISPATCH_INTEGRAL_TYPES(A.type(), "pscan_transpose_cuda", ([&]
    //                                                               {
    //     for (size_t i = 0; i < num_streams; ++i){
    //         dim3 dimGrid(dim_size/TILE_DIM, state_size/TILE_DIM, batch_size / num_streams); // (8. 64, 96),  256 / 32, 2048 / 32, 384 / 4
    //         dim3 dimBlock(TILE_DIM, BLOCK_ROWS, 1); // 32, 8, 1
    //         transposeNoBankConflicts<<<dimGrid, dimBlock, 0, streams[i]>>>(
    //             X_.data<scalar_t>()+ offset*i,
    //             X.data<scalar_t>() + offset*i,
    //             X.stride(0)
    //         );
    //     } }));

    // const int threads = 1024;
    // const int elements_per_thread = 2;

    // AT_DISPATCH_INTEGRAL_TYPES(A.type(), "pscan_forward_cuda", ([&]
    //                                                             {
    // for(size_t i = 0; i < num_streams; ++i){
    //     const auto blocks = dim3(batch_size / num_streams, dim_size, 1);

    //     typedef typename PairScalar<int8_t>::type pair_type;
    //     auto block_scan_temp_bytes = archs_max_bytes<pair_type, threads, 700, 800, 860>;
    //     auto smem_size = (std::max)(1 * sizeof(pair_type), block_scan_temp_bytes);
    
    //     pscan_cuda_forward_kernel<int8_t, elements_per_thread, threads, REVERSE><<<blocks, threads, smem_size, streams[i]>>>(
    //         A.data<int8_t>() + offset*i,
    //         X_.data<int8_t>() + offset*i,
    //         dim_size,
    //         state_size
    //     );
    // } }));

    for (size_t i = 0; i < num_streams; ++i)
    {
        CHECK_CUDA_ERROR(hipStreamSynchronize(streams[i]));
        CHECK_CUDA_ERROR(hipStreamDestroy(streams[i]));
    }

    return X_;
}

template torch::Tensor pscan_cuda_wrapper<true>(torch::Tensor A, torch::Tensor X);
template torch::Tensor pscan_cuda_wrapper<false>(torch::Tensor A, torch::Tensor X);
